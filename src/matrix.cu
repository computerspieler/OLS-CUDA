#include "hip/hip_runtime.h"
#include "matrix.cuh"

#include <cstdio>

#include ""
#include "hip/hip_runtime.h"

// CTI: Coordonates To Index
#define CTI(i, j) ((i) * N + (j))

#define CHECK_CUDA_ERROR(msg) { \
	if (status != hipSuccess) { \
		fprintf(stderr, "[" __FILE__ ":%d]" msg "\n", __LINE__, hipGetErrorString(status)); \
		return status; \
	} \
}

#define TRY_KERNEL(...) { \
	__VA_ARGS__; \
	status = hipGetLastError(); \
	CHECK_CUDA_ERROR("Kernel launch failed: %s"); \
}

__global__ void setToIdentity(
	float* output,
	int N
)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= N || j >= N)
		return;

	output[CTI(i, j)] = (i == j) ? 1 : 0;
}

__global__ void getMaxValue(
	float* mat,
	float* max_val,
	int* row_id,
	int N,
	int i,
	int j
) {
	if(j < i)
		return;

	float val = mat[CTI(j, i)];
	
	if(abs(val) > abs(*max_val)) {
		*max_val = val;
		*row_id = j;
	}
}

__global__ void swapRows(
	float* mat,
	int* row_id,
	int N,
	int k
) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= N)
		return;

	float tmp = mat[CTI(k, j)];
	mat[CTI(k, j)] = mat[CTI(*row_id, j)];
	mat[CTI(*row_id, j)] = tmp;
}

__global__ void divideRow(
	float* mat_out,
	float* mat_tmp,
	float* max_val,
	int N,
	int k
) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= N)
		return;

	mat_tmp[CTI(k, j)] /= *max_val;
	mat_out[CTI(k, j)] /= *max_val;
}

__global__ void substractRows(
	float* mat_out,
	float* mat_tmp,
	int N,
	int k
) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= N || i == k)
		return;

	if (j >= N)
		return;

	float v1 = mat_tmp[CTI(i, k)] * mat_out[CTI(k, j)];
	float v2 = mat_tmp[CTI(i, k)] * mat_tmp[CTI(k, j)];

	mat_out[CTI(i, j)] -= v1;

	if(j > k)
		mat_tmp[CTI(i, j)] -= v2;
}

hipError_t compute_inverse_matrix(
	float *input,
	float *tmp,
	float *output,
	const int N
) {
	hipError_t status;
	float *max_val;
	int *max_val_row;

	status = hipMalloc((void**)&max_val, sizeof(float));
	CHECK_CUDA_ERROR("hipMalloc failed! %s");

	status = hipMalloc((void**)&max_val_row, sizeof(int));
	CHECK_CUDA_ERROR("hipMalloc failed! %s");

	status = hipMemcpy(tmp, input, N * N * sizeof(float), hipMemcpyDeviceToDevice);
	CHECK_CUDA_ERROR("hipMemcpy failed! %s");

	// Compute the inverted matrix
	setToIdentity << <dim3((N + 31) / 32, (N + 31) / 32), dim3(32, 32) >> > (
		output, N
	);
	status = hipGetLastError();
	CHECK_CUDA_ERROR("kernel launch failed: %s");

	for (int k = 0; k < N; k++) {
		status = hipMemset(max_val, 0, sizeof(float));
		CHECK_CUDA_ERROR("hipMemset failed! %s");
		status = hipMemset(max_val_row, 0, sizeof(int));
		CHECK_CUDA_ERROR("hipMemset failed! %s");

		status = hipDeviceSynchronize();
		CHECK_CUDA_ERROR("An error occured while executing hipDeviceSynchronize: %s");

		for(int j = k; j < N; j ++) {
			TRY_KERNEL(
				getMaxValue << <dim3(1, 1), dim3(1, 1) >> > (
					tmp, max_val, max_val_row, N, k, j
				)
			);
		}
		
		TRY_KERNEL(
			swapRows << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				tmp, max_val_row, N, k
			)
		);

		TRY_KERNEL(
			swapRows << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				output, max_val_row, N, k
			)
		);
		
		TRY_KERNEL(
			divideRow << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				output, tmp, max_val, N, k
			)
		);
		
		TRY_KERNEL(
			substractRows << <dim3((N + 31) / 32, (N + 31) / 32), dim3(32, 32) >> > (
				output, tmp, N, k
			)
		);
		status = hipDeviceSynchronize();
		CHECK_CUDA_ERROR("An error occured while executing hipDeviceSynchronize: %s");
	}

	status = hipDeviceSynchronize();
	CHECK_CUDA_ERROR("An error occured while executing hipDeviceSynchronize: %s");

	status = hipFree(max_val);
	CHECK_CUDA_ERROR("hipFree failed! %s");

	status = hipFree(max_val_row);
	CHECK_CUDA_ERROR("hipFree failed! %s");

	return status;
}
