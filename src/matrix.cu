#include "hip/hip_runtime.h"
#include "matrix.cuh"

#include <cstdio>

#include ""
#include "hip/hip_runtime.h"

// CTI: Coordonates To Index
#define CTI(i, j) ((i) * N + (j))

#define CHECK_CUDA_ERROR(msg) { \
	if (status != hipSuccess) { \
		fprintf(stderr, "[" __FILE__ ":%d]" msg "\n", __LINE__, hipGetErrorString(status)); \
		return status; \
	} \
}

#define TRY_KERNEL(...) { \
	__VA_ARGS__; \
	status = hipGetLastError(); \
	CHECK_CUDA_ERROR("Kernel launch failed: %s"); \
}

__global__ void setToIdentity(
	float* output,
	int N
)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= N || j >= N)
		return;

	output[CTI(i, j)] = (i == j) ? 1 : 0;
}

__global__ void getMaxValue(
	float* mat,
	float* max_val,
	int* col_id,
	int N,
	int i
) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	float val = abs(mat[CTI(i, j)]);
	if(val > *max_val) {
		*max_val = val;
		*col_id = i;
	}
}

__global__ void swapRows(
	float* mat,
	int* col_id,
	int N,
	int k
) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= N)
		return;

	float tmp = mat[CTI(k, j)];
	mat[CTI(k, j)] = mat[CTI(*col_id, j)];
	mat[CTI(*col_id, j)] = tmp;
}

__global__ void divideRow(
	float* mat,
	float* max_val,
	int N,
	int k
) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= N)
		return;

	mat[CTI(k, j)] /= *max_val;
}

hipError_t compute_inverse_matrix(
	float *input,
	float *tmp,
	float *output,
	const int N
) {
	hipError_t status;
	float *max_val;
	int *max_val_col;

	status = hipMalloc((void**)&max_val, sizeof(float));
	CHECK_CUDA_ERROR("hipMalloc failed! %s");

	status = hipMalloc((void**)&max_val_col, sizeof(int));
	CHECK_CUDA_ERROR("hipMalloc failed! %s");

	status = hipMemcpy(tmp, input, N * N * sizeof(float), hipMemcpyDeviceToDevice);
	CHECK_CUDA_ERROR("hipMemcpy failed! %s");

	// Compute the inverted matrix
	setToIdentity << <dim3((N + 31) / 32, (N + 31) / 32), dim3(32, 32) >> > (
		output, N
	);
	status = hipGetLastError();
	CHECK_CUDA_ERROR("kernel launch failed: %s");

	for (int k = 0; k < N; k++) {
		status = hipMemset(max_val, 0, sizeof(float));
		CHECK_CUDA_ERROR("hipMemset failed! %s");

		TRY_KERNEL(
			getMaxValue << <dim3(N, 1), dim3(1, 1) >> > (
				tmp, max_val, max_val_col, N, k
			)
		);

		status = hipDeviceSynchronize();
		CHECK_CUDA_ERROR("An error occured while executing hipDeviceSynchronize: %s");
		
		TRY_KERNEL(
			swapRows << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				tmp, max_val_col, N, k
			)
		);
		TRY_KERNEL(
			divideRow << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				tmp, max_val, N, k
			)
		);
		
		TRY_KERNEL(
			swapRows << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				output, max_val_col, N, k
			)
		);
		TRY_KERNEL(
			divideRow << <dim3((N + 31) / 32, 1), dim3(32, 1) >> > (
				output, max_val, N, k
			)
		);

		
	}

	status = hipDeviceSynchronize();
	CHECK_CUDA_ERROR("An error occured while executing hipDeviceSynchronize: %s");

	status = hipFree(max_val);
	CHECK_CUDA_ERROR("hipFree failed! %s");

	status = hipFree(max_val_col);
	CHECK_CUDA_ERROR("hipFree failed! %s");

	return status;
}
